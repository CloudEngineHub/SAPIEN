#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

namespace {

struct PxBodyData {
  float4 q;
  float3 p;
  float padding0;
  float3 v;
  float padding1;
  float3 w;
  float padding2;
};
static_assert(sizeof(PxBodyData) == 64);

struct SapienBodyData {
  float pq[7];
  float3 v;
  float3 w;
  float padding0;
  float padding1;
  float padding2;
};
static_assert(sizeof(SapienBodyData) == 64);

struct PxTransformWrong {
  float4 q;
  float3 p;
};
static_assert(sizeof(PxTransformWrong) == 32);

struct PxTransform {
  float q[4];
  float p[3];
};
static_assert(sizeof(PxTransform) == 28);

struct SapienTransform {
  float pq[7];
};
static_assert(sizeof(SapienTransform) == 28);

} // namespace

__global__ void transform_sapien_to_physx_add_offset(SapienTransform *__restrict__ data,
                                                     int const *__restrict__ index,
                                                     float4 const *__restrict__ offset,
                                                     int count) {
  int g = blockIdx.x * blockDim.x + threadIdx.x;
  if (g >= count) {
    return;
  }

  int i = index[g];

  union {
    PxTransform pd{};
    SapienTransform sd;
  };

  pd.q[0] = data[i].pq[4];
  pd.q[1] = data[i].pq[5];
  pd.q[2] = data[i].pq[6];
  pd.q[3] = data[i].pq[3];

  pd.p[0] = data[i].pq[0] + offset[i].x;
  pd.p[1] = data[i].pq[1] + offset[i].y;
  pd.p[2] = data[i].pq[2] + offset[i].z;

  data[i] = sd;
}

__global__ void transform_physx_to_sapien_subtract_offset(PxTransform *__restrict__ data,
                                                          int const *__restrict__ index,
                                                          float4 const *__restrict__ offset,
                                                          int count) {
  int g = blockIdx.x * blockDim.x + threadIdx.x;
  if (g >= count) {
    return;
  }
  int i = index[g];

  union {
    SapienBodyData sd{};
    PxTransform pd;
  };

  sd.pq[0] = data[i].p[0] - offset[i].x;
  sd.pq[1] = data[i].p[1] - offset[i].y;
  sd.pq[2] = data[i].p[2] - offset[i].z;

  sd.pq[3] = data[i].q[3];
  sd.pq[4] = data[i].q[0];
  sd.pq[5] = data[i].q[1];
  sd.pq[6] = data[i].q[2];

  data[i] = pd;
}

__global__ void body_data_sapien_to_physx_add_offset(SapienBodyData *__restrict__ data,
                                                     int4 const *__restrict__ index,
                                                     float4 const *__restrict__ offset,
                                                     int count) {
  int g = blockIdx.x * blockDim.x + threadIdx.x;
  if (g >= count) {
    return;
  }
  int i = index[g].x;

  union {
    PxBodyData pd{};
    SapienBodyData sd;
  };

  pd.q = make_float4(data[i].pq[4], data[i].pq[5], data[i].pq[6], data[i].pq[3]);
  pd.p = make_float3(data[i].pq[0] + offset[i].x, data[i].pq[1] + offset[i].y,
                     data[i].pq[2] + offset[i].z);
  pd.v = data[i].v;
  pd.w = data[i].w;

  data[i] = sd;
}

__global__ void body_data_physx_to_sapien_subtract_offset(PxBodyData *__restrict__ data,
                                                          int4 const *__restrict__ index,
                                                          float4 const *__restrict__ offset,
                                                          int count) {
  int g = blockIdx.x * blockDim.x + threadIdx.x;
  if (g >= count) {
    return;
  }
  int i = index[g].x;

  union {
    SapienBodyData sd{};
    PxBodyData pd;
  };

  sd.pq[0] = data[i].p.x - offset[i].x;
  sd.pq[1] = data[i].p.y - offset[i].y;
  sd.pq[2] = data[i].p.z - offset[i].z;

  sd.pq[3] = data[i].q.w;
  sd.pq[4] = data[i].q.x;
  sd.pq[5] = data[i].q.y;
  sd.pq[6] = data[i].q.z;

  sd.v = data[i].v;
  sd.w = data[i].w;

  data[i] = pd;
}

namespace sapien {

constexpr int BLOCK_SIZE = 128;

void body_data_physx_to_sapien_subtract_offset(void *data, void *index, void *offset, int count,
                                               void *stream) {
  body_data_physx_to_sapien_subtract_offset<<<(count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0,
                                              (hipStream_t)stream>>>(
      (PxBodyData *)data, (int4 *)index, (float4 *)offset, count);
}

void body_data_sapien_to_physx_add_offset(void *data, void *index, void *offset, int count,
                                          void *stream) {
  body_data_sapien_to_physx_add_offset<<<(count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0,
                                         (hipStream_t)stream>>>(
      (SapienBodyData *)data, (int4 *)index, (float4 *)offset, count);
}

void transform_sapien_to_physx_add_offset(void *data, void *index, void *offset, int count,
                                          void *stream) {
  transform_sapien_to_physx_add_offset<<<(count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0,
                                         (hipStream_t)stream>>>(
      (SapienTransform *)data, (int *)index, (float4 *)offset, count);
}

void transform_physx_to_sapien_subtract_offset(void *data, void *index, void *offset, int count,
                                               void *stream) {
  transform_physx_to_sapien_subtract_offset<<<(count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0,
                                              (hipStream_t)stream>>>(
      (PxTransform *)data, (int *)index, (float4 *)offset, count);
}

}; // namespace sapien
